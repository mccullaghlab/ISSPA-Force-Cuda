#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_vector_routines.h"
#include "angle_class.h"
#include "angle_force_cuda.h"
#include "constants.h"

// CUDA Kernels

__global__ void angle_force_kernel(float4 *xyz, float4 *f, int nAtoms, float lbox, int4 *angleAtoms, float2 *angleParams, int nAngles) {
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	int4 atoms;
	float4 r1;
	float4 r2;
	float c11, c22, c12;
	float b;
	float theta;
	float fang;
	float hbox;
	float2 params;
	

	if (index < nAngles)
	{
		hbox = 0.5f*lbox;
		// determine atoms to work on
		atoms = __ldg(angleAtoms+index);
		// get distance vectors separating the atoms
		r1 = min_image(__ldg(xyz+atoms.x) - __ldg(xyz+atoms.y),lbox,hbox);
		r2 = min_image(__ldg(xyz+atoms.y) - __ldg(xyz+atoms.z),lbox,hbox);
		// compute dot products
		c11 = r1.x*r1.x + r1.y*r1.y + r1.z*r1.z;
		c22 = r2.x*r2.x + r2.y*r2.y + r2.z*r2.z;
		c12 = r1.x*r2.x + r1.y*r2.y + r1.z*r2.z;
		b = -__fdividef(c12,sqrtf(c11*c22));
		// make sure b is in the domain of the arccos
		if (b>=1.0f) {
			// theta is zero
			theta = 1.0e-16f;
		} else if (b <= -1.0f) {
			// theta is pi
			theta = PI;
		} else {
			// b is in domain so take arccos
			theta = acosf(b);
		}
		// grab parameters for angle atoms type - stored as fourth integer in angleAtoms
		params = __ldg(angleParams+atoms.w);
		// compute force component
		fang = __fdividef(params.x*(theta - params.y),sqrtf(c11*c22-c12*c12));
		// atomicAdd forces to each atom
		atomicAdd(&(f[atoms.x].x), fang*(c12/c11*r1.x-r2.x));
		atomicAdd(&(f[atoms.y].x), fang*((1.0f+c12/c22)*r2.x-(1.0f+c12/c11)*r1.x));
		atomicAdd(&(f[atoms.z].x), fang*(r1.x-c12/c22*r2.x));
		atomicAdd(&(f[atoms.x].y), fang*(c12/c11*r1.y-r2.y));
		atomicAdd(&(f[atoms.y].y), fang*((1.0f+c12/c22)*r2.y-(1.0f+c12/c11)*r1.y));
		atomicAdd(&(f[atoms.z].y), fang*(r1.y-c12/c22*r2.y));
		atomicAdd(&(f[atoms.x].z), fang*(c12/c11*r1.z-r2.z));
		atomicAdd(&(f[atoms.y].z), fang*((1.0f+c12/c22)*r2.z-(1.0f+c12/c11)*r1.z));
		atomicAdd(&(f[atoms.z].z), fang*(r1.z-c12/c22*r2.z));

	}
}

/* C wrappers for kernels */

float angle_force_cuda(float4 *xyz_d, float4 *f_d, int nAtoms, float lbox, angle& angles) 
{
	float milliseconds;
	// initialize timing stuff
	hipEventRecord(angles.angleStart);
	
	// run angle cuda kernel
	angle_force_kernel<<<angles.gridSize, angles.blockSize>>>(xyz_d, f_d, nAtoms, lbox, angles.angleAtoms_d, angles.angleParams_d, angles.nAngles);

	// finalize timing
	hipEventRecord(angles.angleStop);
	hipEventSynchronize(angles.angleStop);
	hipEventElapsedTime(&milliseconds, angles.angleStart, angles.angleStop);

	// return time
	return milliseconds;

}

void angle_force_cuda_grid_block(int nAngles, int *gridSize, int *blockSize, int *minGridSize)
{
	// determine gridSize and blockSize
	hipOccupancyMaxPotentialBlockSize(minGridSize, blockSize, angle_force_kernel, 0, nAngles); 

    	// Round up according to array size 
    	*gridSize = (nAngles + *blockSize - 1) / *blockSize; 
}
