#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "cuda_vector_routines.h"
#include "isspa_class.h"
#include "isspa_force_cuda.h"
#include "constants.h"
#include "hip/hip_runtime_api.h"

using namespace std;

// constants
__constant__ int nTypes;
__constant__ int nMC;
__constant__ int nRs;
__constant__ int nGRs;
__constant__ int nERs;
__constant__ int nAtoms;
__constant__ int nPairs;
__constant__ float2 box;
__constant__ float2 forceRparams;
__constant__ float2 gRparams;
__constant__ float2 eRparams;

// device functions

// CUDA Kernels

__device__ float atomicMul(float* address, float val) { 
        unsigned int* address_as_u = (unsigned int*)address; 
        unsigned int old = *address_as_u, assumed; 
        do { 
	        assumed = old; 
		old = atomicCAS(address_as_u, assumed, __float_as_uint(val * __uint_as_float(assumed))); 
	} while (assumed != old); return __uint_as_float(old);
}

__inline__ __device__ float warpReduceMul(float val) {
  for (int offset = warpSize/2; offset > 0; offset /= 2)
          val *= __shfl_down(val, offset);
  return val;
}

__inline__ __device__
float4 warpReduceSumQuad(float4 val) {
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
                val.x += __shfl_down(val.x, offset);
                val.y += __shfl_down(val.y, offset);
                val.z += __shfl_down(val.z, offset);
                val.w += __shfl_down(val.w, offset);
        }
        return val; 
}

__inline__ __device__
float4 warpReduceSumTriple(float4 val) {
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
                val.x += __shfl_down(val.x, offset);
                val.y += __shfl_down(val.y, offset);
                val.z += __shfl_down(val.z, offset);
        }
        return val; 
}

__global__  void isspa_MC_points_kernel(float4 *xyz, float4 *mcpos, hiprandState *state, float *rmax, int *isspaTypes) {
        unsigned int MC = threadIdx.x + blockIdx.x*blockDim.x;    
	int atom = blockIdx.x;
	int it;
	float r2;
	float rmax_l;
	float4 mcr;
	float4 mcpos_l;
	hiprandState_t threadState;
	
	// Determine which atom the MC point is being generated on
	it = __ldg(isspaTypes+atom);
	rmax_l = __ldg(rmax+it);
	mcpos_l = __ldg(xyz+atom);
	threadState = state[MC];	
	do {
	        mcr.x = (2.0f * hiprand_uniform(&threadState) - 1.0f);
		mcr.y = (2.0f * hiprand_uniform(&threadState) - 1.0f);
		mcr.z = (2.0f * hiprand_uniform(&threadState) - 1.0f);
		r2 = mcr.x*mcr.x + mcr.y*mcr.y + mcr.z*mcr.z;
	}
	while (r2 >= 1.0f);
	mcr *= rmax_l;
	mcpos_l += mcr;
	mcpos_l.w = 1.0;
	mcpos[MC] = mcpos_l;
}



__global__ void isspa_field_kernel(float4 *xyz, float *rmax, int *isspaTypes, float *gTable, float *eTable, float4 *enow, float4 *e0now, float4 *mcpos, int nThreads) { 
        unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	int atom;
        int atom2;
	int MC;
	int MCind;
	int bin;
        int it;
	int jt;
        float rmax_l;
        float dist2, dist;
        float fracDist;
        float g1, g2;
        float e1, e2;
        float etab;
        float2 gRparams_l = gRparams;
        float2 eRparams_l = eRparams;
	float4 atom2_pos;
        float4 r;
        float4 mcpos_l;
        float4 enow_l;
        float4 e0now_l;
	// Determine which atom the MC point is being generated on
	atom = int(index/(float) (nThreads*nMC));
	MC = int(index /(float) (nThreads));
	MCind = int(MC - atom*nMC);
	atom2 = int(index - atom*nMC*nThreads - MCind*nThreads);
	if (atom < nAtoms) {
	        if (MCind < nMC) {
		        if (atom2 < nAtoms) {
			        // Get atom positions
			        mcpos_l = __ldg(mcpos+MC);
				it = __ldg(isspaTypes+atom);
				rmax_l = __ldg(rmax+it);

                                // Set e0now to zero
                                enow_l.x = 0.0;
                                enow_l.y = 0.0;
                                enow_l.z = 0.0;	  
                                enow_l.w = 0.0;	  

				// Get atom positions
				atom2_pos = __ldg(xyz+atom2);
				// Get constants for atom
				jt = __ldg(isspaTypes+atom2);
				r = min_image(mcpos_l - atom2_pos,box.x,box.y);
				dist2 = r.x*r.x + r.y*r.y + r.z*r.z;
				dist = sqrtf(dist2);			
				if (dist <= rmax_l) {
				        e0now_l.w = 1;
					// determine density bin of distance
					bin = int ( (dist-gRparams_l.x)/gRparams_l.y ); 	
					// make sure bin is in limits of density table
					if (bin < 0) {
					        mcpos_l.w = 0.0;
					} else if (bin < nGRs) {
					        // Push Density to MC point
					        fracDist = (dist - (gRparams_l.x+bin*gRparams_l.y)) / gRparams_l.y; 	        
						g1 = __ldg(gTable+jt*nGRs+bin);
						g2 = __ldg(gTable+jt*nGRs+bin+1);
						mcpos_l.w = g1*(1.0-fracDist)+g2*fracDist;
						// Push electric field to MC point
						fracDist = (dist - (eRparams_l.x+bin*eRparams_l.y)) / eRparams_l.y;
						e1 = __ldg(eTable+jt*nERs+bin);
						e2 = __ldg(eTable+jt*nERs+bin+1);					
						etab =  e1*(1.0-fracDist)+e2*fracDist;
						enow_l += r/dist*etab;
                                        }      
				} else {
				        e0now_l = -e0*atom2_pos.w*r/dist2/dist;
					e0now_l.w = 0.0;
					mcpos_l.w = 1.0;						
				}				
				enow_l -= e0*atom2_pos.w*r/dist2/dist;
                                
                        } else {
                                enow_l.x = 0.0;
                                enow_l.y = 0.0;
                                enow_l.z = 0.0;
                                e0now_l.x = 0.0;
                                e0now_l.y = 0.0;
                                e0now_l.z = 0.0;	  
                                e0now_l.w = 0.0;	  
                                mcpos_l.w = 1.0;
			}

                     
			mcpos_l.w = warpReduceMul(mcpos_l.w);	
			enow_l =  warpReduceSumTriple(enow_l);
			e0now_l =  warpReduceSumQuad(e0now_l);
			
			if ((threadIdx.x & (warpSize - 1)) == 0) {
			        atomicMul(&(mcpos[MC].w), mcpos_l.w);
				atomicAdd(&(enow[MC].x), enow_l.x);
				atomicAdd(&(enow[MC].y), enow_l.y);
				atomicAdd(&(enow[MC].z), enow_l.z);
                                atomicAdd(&(e0now[MC].x), e0now_l.x);
				atomicAdd(&(e0now[MC].y), e0now_l.y);
				atomicAdd(&(e0now[MC].z), e0now_l.z);
				atomicAdd(&(e0now[MC].w), e0now_l.w);
			}				
		}
	}	
}

__global__ void isspa_force_kernel(float4 *xyz, float *vtot, float *rmax, int *isspaTypes, float *forceTable, float4 *f, hiprandState *state,  float4 *enow, float4 *e0now, float4 *mcpos, float nThreads, float4 *isspaf) {
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	int bin;
        int jt;
	int MC;
	int atom;
	float igo;
	float fs;
	float r2;
	float r0;
        float rmax_l;
	float vtot_l;
        float dist2, dist;
        float pdotr;
	float cothE;
	//float c1,c2,c3;
	//float dp1,dp2,dp3;
	float Rz;
	float4 xyz_l;
        float4 r;
        float4 fi;
        //float4 fj;
	float4 mcpos_l;
	float4 enow_l;
	float4 e0now_l;

	// Determine the atom for which the force is being summed on
	atom = int(index/(float) (nThreads));
	MC = int((index-atom*nThreads));
        
	// Zero out the forces
	fi.x = 0.0;
	fi.y = 0.0;
	fi.z = 0.0;
	//fj.x = 0.0;
	//fj.y = 0.0;
	//fj.z = 0.0;
                
	if (MC < nAtoms*nMC) {
        
	        // Load in position, atom type, and rmax of atom
	        xyz_l = __ldg(xyz+atom);
		jt = __ldg(isspaTypes + atom);
		rmax_l = __ldg(rmax+jt);	
		vtot_l = __ldg(vtot+jt);
		
		// Load in field data for the MC point	
		mcpos_l = __ldg(mcpos+MC);
		enow_l = __ldg(enow+MC);
		e0now_l = __ldg(e0now+MC);                


		igo = vtot_l/e0now_l.w;
                mcpos_l.w *= igo;

                
		// Convert enow into polarzation
		r2 = enow_l.x*enow_l.x+enow_l.y*enow_l.y+enow_l.z*enow_l.z;
		r0 = sqrtf(r2);


		enow_l /= r0;			
		enow_l.w = r0;
		e0now_l /= 3.0;
		e0now_l.w = igo;


                
		// Calculate the distance between the MC point and atom1
		r = min_image(mcpos_l - xyz_l,box.x,box.y);
		dist2 = r.x*r.x + r.y*r.y + r.z*r.z;
		dist = sqrtf(dist2);
                

		// Coulombic Force
		cothE=1.0/tanhf(enow_l.w);
		//c1=cothE-1.0/enow_l.w;
		//c2=1.0-2.0*c1/enow_l.w;
		//c3=cothE-3.0*c2/enow_l.w;
                
		Rz=(enow_l.x*r.x+enow_l.y*r.y+enow_l.z*r.z)/dist;
		//dp1=3.0*Rz;
		//dp2=7.5*Rz*Rz-1.5;
		//dp3=(17.50*Rz*Rz-7.50)*Rz;                

                
                //fs = -xyz_l.w*p0*c1/dist2/dist*mcpos_l.w;
                //fi += fs*(dp1*r/dist-enow_l);
		fs = -xyz_l.w*p0*(cothE-1.0/enow_l.w)/dist2/dist*mcpos_l.w;
                fi += fs*((3.0*Rz)*r/dist-enow_l);
		//fj += fs*((3.0*Rz)*r/dist-enow_l);
                //fs = -xyz_l.w*q0*(1.5*c2-0.5)/dist2/dist2*mcpos_l.w;
                //fi += fs*(dp2*r/dist-dp1*enow_l);
		fs = -xyz_l.w*q0*(1.5*(1.0-2.0*(cothE-1.0/enow_l.w)/enow_l.w)-0.5)/dist2/dist2*mcpos_l.w;
                fi += fs*((7.5*Rz*Rz-1.5)*r/dist-(3.0*Rz)*enow_l);
		//fj += fs*((7.5*Rz*Rz-1.5)*r/dist-(3.0*Rz)*enow_l);
		//fs = -xyz_l.w*o0*(2.5*c3-1.5*c1)/dist2/dist2/dist*mcpos_l.w;
                //fi += fs*(dp3*r/dist-dp2*enow_l);
		fs = -xyz_l.w*o0*(2.5*(cothE-3.0*(1.0-2.0*(cothE-1.0/enow_l.w)/enow_l.w)/enow_l.w)-1.5*(cothE-1.0/enow_l.w))/dist2/dist2/dist*mcpos_l.w;
                fi += fs*(((17.50*Rz*Rz-7.50)*Rz)*r/dist-(7.5*Rz*Rz-1.5)*enow_l);
		//fj += fs*(((17.50*Rz*Rz-7.50)*Rz)*r/dist-(7.5*Rz*Rz-1.5)*enow_l);

		// Lennard-Jones Force  
		if (dist <= rmax_l) {
		        bin = int ( (dist-forceRparams.x)/forceRparams.y + 0.5f);
			if (bin >= (nRs)) {
			        fs = 0.0;
			} else {
			        //Lennard-Jones Force 
			        //fracDist = (dist - (forceRparams.x+bin*forceRparams.y)) / forceRparams.y;
			        //f1 = __ldg(forceTable+it*nRs+bin);
			        //f2 = __ldg(forceTable+it*nRs+bin+1);
			        //fs = (f1*(1.0-fracDist)+f2*fracDist)*mcpos_l.w;
			        fs = __ldg(forceTable + jt*nRs+bin)*mcpos_l.w;
			}
			fi += -fs*r/dist;
			//fj += -fs*r/dist;                        
		} else {
		        // Constant Density Dielectric
		        fs=-xyz_l.w*p0/dist2/dist;
			pdotr=3.0*(e0now_l.x*r.x+e0now_l.y*r.y+e0now_l.z*r.z)/dist2;
			fi += fs*(pdotr*r-e0now_l)*e0now_l.w;
			//fj += fs*(pdotr*r-e0now_l)*e0now_l.w;			
		}	
	}
        
	fi =  warpReduceSumTriple(fi);
	//fj =  warpReduceSumTriple(fj);
	
	if ((threadIdx.x & (warpSize - 1)) == 0) {
                atomicAdd(&(f[atom].x), fi.x);
                atomicAdd(&(f[atom].y), fi.y);
                atomicAdd(&(f[atom].z), fi.z);
                //atomicAdd(&(isspaf[atom].x), fj.x);
                //atomicAdd(&(isspaf[atom].y), fj.y);
                //atomicAdd(&(isspaf[atom].z), fj.z);
	}
}


/* C wrappers for kernels */
float isspa_force_cuda(float4 *xyz_d, float4 *f_d, float4 *isspaf_d, isspa& isspas, int nAtoms_h) {
        //float isspa_force_cuda(float4 *xyz_d, float4 *f_d, isspa& isspas, int nAtoms_h) {
        
        float milliseconds;
        
        // timing                                                                                                                
        hipEventRecord(isspas.isspaStart);
        
	hipProfilerStart();
        
	// compute position of each MC point
	isspa_MC_points_kernel<<<nAtoms_h,isspas.nMC >>>(xyz_d, isspas.mcpos_d, isspas.randStates_d, isspas.rmax_d, isspas.isspaTypes_d);
        // compute densities and mean electric field value for each MC point
	isspa_field_kernel<<<isspas.mcGridSize, isspas.mcBlockSize>>>(xyz_d, isspas.rmax_d, isspas.isspaTypes_d, isspas.isspaGTable_d, isspas.isspaETable_d, isspas.enow_d, isspas.e0now_d, isspas.mcpos_d, isspas.mcThreads);
	// compute forces for each atom
	isspa_force_kernel<<<isspas.fGridSize, isspas.fBlockSize>>>(xyz_d,isspas.vtot_d,isspas.rmax_d,isspas.isspaTypes_d,isspas.isspaForceTable_d,f_d,isspas.randStates_d,isspas.enow_d,isspas.e0now_d,isspas.mcpos_d,isspas.fThreads,isspaf_d);
        
	hipDeviceSynchronize();
	hipProfilerStop();
	
        // finish timing
	hipEventRecord(isspas.isspaStop);
	hipEventSynchronize(isspas.isspaStop);
	hipEventElapsedTime(&milliseconds, isspas.isspaStart, isspas.isspaStop);
	return milliseconds;
}

void isspa_grid_block(int nAtoms_h, int nPairs_h, float lbox_h, isspa& isspas) {
        
        float2 box_h;
	int maxThreadsPerBlock = 1024;
	int temp;
        
	// determine gridSize and blockSize for field kernel	
	temp = int(ceil((nAtoms_h) / (float) 32.0));
	isspas.mcThreads = temp*32;		
	isspas.mcGridSize = int(ceil(isspas.mcThreads*nAtoms_h*isspas.nMC / (float) maxThreadsPerBlock));
	isspas.mcBlockSize = maxThreadsPerBlock;
        
	printf("Number of field kernel blocks: %d \n", isspas.mcGridSize);
	printf("Number of field kernel threads per block: %d \n", isspas.mcBlockSize);
	printf("Number of field kernel threads per MC point: %d \n", isspas.mcThreads);
	
        // determine gridSize and blockSize for force kernel
	temp = int(ceil((nAtoms_h*isspas.nMC) / (float) 32.0));
	isspas.fThreads = temp*32;		
	isspas.fGridSize = int(ceil(isspas.fThreads*nAtoms_h / (float) maxThreadsPerBlock));
	isspas.fBlockSize = maxThreadsPerBlock;
        
	printf("Number of force kernel blocks: %d \n", isspas.fGridSize);
	printf("Number of force kernel threads per block: %d \n", isspas.fBlockSize);
	printf("Number of force kernel ISSPA MC-atom pair calculations per thread: %d \n", isspas.fThreads);
	
	// fill box with box and half box length
	box_h.x = lbox_h;
	box_h.y = lbox_h/2.0;
	
	// set constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(nMC), &isspas.nMC, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nTypes), &isspas.nTypes, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nRs), &isspas.nRs, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nGRs), &isspas.nGRs, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nERs), &isspas.nERs, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nAtoms), &nAtoms_h, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nPairs), &nPairs_h, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(box), &box_h, sizeof(float2));
	hipMemcpyToSymbol(HIP_SYMBOL(forceRparams), &isspas.forceRparams, sizeof(float2));
	hipMemcpyToSymbol(HIP_SYMBOL(gRparams), &isspas.gRparams, sizeof(float2));	
	hipMemcpyToSymbol(HIP_SYMBOL(eRparams), &isspas.eRparams, sizeof(float2));	
}
